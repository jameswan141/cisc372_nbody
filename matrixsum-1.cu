#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include "kernel.h"
#include "kernel.cu"
#include "compute.h"
#include <math.h>

using namespace std;

int main()
{
    int NUMELEMENTS=10;
    int SIZE=NUMELEMENTS*NUMELEMENTS;

    vector<float> h_i(SIZE);
    vector<float> h_j(SIZE);
    vector<float> h_k(SIZE);

    for (int i=0; i<NUMELEMENTS; i++){
        for (int j=0; j<NUMELEMENTS; j++){
            h_a[i*NUMELEMENTS+j]=sin(i);
            h_b[i*NUMELEMENTS+j]=cos(j);
        }
    }

    compute<float> d_a(SIZE);
    compute<float> d_b(SIZE);
    compute<float> d_c(SIZE);

    d_a.set(&h_a[0], SIZE);
    d_b.set(&h_b[0], SIZE);

    matrixSum(d_a.getData(), d_b.getData(), d_c.getData(), NUMELEMENTS);
    hipDeviceSynchronize();

    d_c.get(&h_c[0], SIZE);
    hipDeviceSynchronize();

    float *cpu_c;
    cpu_c=new float[SIZE];

    float sum;
    for (int row=0; row<NUMELEMENTS; row++){
        for (int col=0; col<NUMELEMENTS; col++){
            sum=0.f;
            for (int n=0; n<NUMELEMENTS; n++){
                sum+=h_a[row*NUMELEMENTS+n]*h_b[n*NUMELEMENTS+col];
            }
            cpu_c[row*NUMELEMENTS+col]=sum;
        }
    }

    double err=0;
    for (int ROW=0; ROW<NUMELEMENTS; ROW++){
        for (int COL=0; COL<NUMELEMENTS; COL++){
            err+=cpu_c[ROW*NUMELEMENTS+COL]-h_c[ROW*NUMELEMENTS+COL];
        }
    }

    cout << "Error: " << err << endl;

    return 0;
}
