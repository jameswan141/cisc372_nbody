#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include "hip/hip_runtime.h"
#include "kernel.h"
#include <stdlib.h>

using namespace std;

__global__ void matrixSumKernel(float* a, float* b, float* c, int NUMELEMENTS){

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

    float tmpSum=0;

    if (ROW<NUMELEMENTS && COL<NUMELEMENTS){
        for (int i=0; i<NUMELEMENTS; i++) {
            tmpSum+=a[ROW*NUMELEMENTS+i]*b[i*NUMELEMENTS+COL];
        }
    }
    c[ROW*NUMELEMENTS+COL]=tmpSum;
}

void matrixSum(float *a, float *b, float *c, int NUMELEMENTS){

    dim3 threadsPerBlock(NUMELEMENTS, NUMELEMENTS);
    dim3 blocksPerGrid(1, 1);
        if (NUMELEMENTS*NUMELEMENTS>512){
            threadsPerBlock.x=512;
            threadsPerBlock.y=512;
            blocksPerGrid.x=ceil(double(NUMELEMENTS)/double(threadsPerBlock.x));
            blocksPerGrid.y=ceil(double(NUMELEMENTS)/double(threadsPerBlock.y));
        }

    matrixSumKernel<<<blocksPerGrid,threadsPerBlock>>>(a, b, c, NUMELEMENTS);
}
